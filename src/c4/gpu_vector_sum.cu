#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_common.h"

#define N 10000

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    
    // allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));    

    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    // display the first 10 results
    printf("First 100 results:\n");
    for (int i = 0; i < 100; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Verify a few more results to ensure correctness
    bool success = true;
    for (int i = 0; i < N; i++) {
        if (c[i] != a[i] + b[i]) {
            success = false;
            break;
        }
    }
    printf("\nVector addition %s!\n", success ? "successful" : "failed");

    // free the memory allocated on the GPU
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c)); 

    return 0;
}